#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define N 1000	// size of vectors

#define T 10000// number of threads per block


__global__ void vecAdd(int *A, int *B, int *C) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	C[i] = A[i] * 10 + B[i];
}
int main(int argc, char **argv) {

	int size = N * T * sizeof(int);
	int  a[N*T], b[N*T], c[N*T], *devA, *devB, *devC;
	for (int i = 0; i < N*T; i++) {
		/*devA[i] = 0;
		devB[i] = 0;
		devC[i] = 0;*/
		a[i] = i;
		b[i] = 1;
		//c[i] = 0;
	}
	hipMalloc((void**)&devA, size);
	hipMalloc((void**)&devB, size);
	hipMalloc((void**)&devC, size);

	hipMemcpy(devA, a, size, hipMemcpyHostToDevice);
	hipMemcpy(devB, b, size, hipMemcpyHostToDevice);

	vecAdd << <T, N >> > (devA, devB, devC);

	hipMemcpy(c, devC, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);

	for (int i = 0; i < N*T; i++) {

		//c[i] = a[i] * 10 + b[i];

		printf("c[%d]= %d\n", i, c[i]);
	}
	return(0);
}
// Helper function for using CUDA to add vectors in parallel.

